#include "hip/hip_runtime.h"
#include "test.h"

// CUDA kernel definition
template<typename Type>
__global__ void myKernel(CudaClass<Type>* CudaTest) {   
    CudaTest -> setvalue(5.111);
    printf("Device Value = %lf\n", CudaTest->getvalue());
}

__global__ void MauMul(MatrixXd& matrix, MatrixXd& vectorMatrix, MatrixXd& result) {   
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%lf\n", matrix(0,0));
    // if (row < (*matrix).rows() && col < (*vectorMatrix).cols()) {
    //     (*result)(row, col) = (*matrix).row(row) * (*vectorMatrix).col(col);
    // }
}


template <typename Type>
__host__ __device__ void CudaClass<Type>::add(Type& x){
    value += x;
}

template <typename Type>
MatrixXd CudaClass<Type>::MatrixMul(MatrixXd& matrix, MatrixXd& vectorMatrix){
    MatrixXd result(matrix.rows(), vectorMatrix.cols());
    MatrixXd *matrix_gpu, *vectorMatrix_gpu, *result_gpu;
    hipMalloc(&matrix_gpu, sizeof(MatrixXd)*matrix.rows()*matrix.cols());
    hipMalloc(&vectorMatrix_gpu, sizeof(vectorMatrix));
    hipMalloc(&result_gpu, sizeof(result));

    hipMemcpy(matrix_gpu, &matrix, sizeof(MatrixXd)*matrix.rows()*matrix.cols(), hipMemcpyHostToDevice);
    hipMemcpy(vectorMatrix_gpu, &vectorMatrix, sizeof(vectorMatrix), hipMemcpyHostToDevice);

    std::cout << matrix << std::endl;

    // Launch kernel
    dim3 blockSize(4, 4);
    dim3 threadperblock((vectorMatrix.cols() + blockSize.x - 1) / blockSize.x, (matrix.rows() + blockSize.y - 1) / blockSize.y);
    MauMul<<<blockSize, threadperblock>>>(*matrix_gpu, *vectorMatrix_gpu, *result_gpu);
    hipDeviceSynchronize();
    
    hipMemcpy(&result, result_gpu, sizeof(result), hipMemcpyDeviceToHost);

    // std::cout << result << std::endl;

    std::cout << sizeof(vectorMatrix) << std::endl;
    std::cout << sizeof(MatrixXd) << std::endl;

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }


    // Free device memory
    hipFree(matrix_gpu);
    hipFree(vectorMatrix_gpu);
    hipFree(result_gpu);

    return result;
}

template class CudaClass<double>;


// template <typename Type>
// void CudaClass<Type>::test(CudaClass<Type> myClass){

//     CudaClass<Type>* class_gpu;
//     hipMalloc(&class_gpu, sizeof(CudaClass<Type>));
//     hipMemcpy(class_gpu, &myClass, sizeof(CudaClass<Type>), hipMemcpyHostToDevice);
    
//     // Launching CUDA kernel with host function pointer
//     myKernel<<<1, 1>>>(class_gpu);
//     hipDeviceSynchronize();
//     hipMemcpy(&myClass, class_gpu, sizeof(CudaClass<Type>), hipMemcpyDeviceToHost);


//     std::cout << "Host Value after using Kernel= " << myClass.getvalue() << std::endl;

//     hipError_t err = hipGetLastError();
//     if (err != hipSuccess) {
//         fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
//     }
// }

// template void test<int>(CudaClass<int>);
// template void test<double>(CudaClass<double>);